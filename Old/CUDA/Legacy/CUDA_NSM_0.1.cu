#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int ITER = 5e2;
const int L = 1024;                    // Lattice size (LxL)
const int N = L * L;
const float J = 1.0f;
const float beta = 0.5f;        // Critical temperature inverse for the 2D Ising model
const int THREADS_PER_BLOCK = 16;     // Assuming the number of threads per block side
const int BLOCKS_PER_GRID = L / THREADS_PER_BLOCK;

// CUDA kernel to setup the random state
__global__ void setupStates(hiprandState* states, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * L + idx;
    if (idx < L && idy < L) {
        hiprand_init(seed, id, 0, &states[id]);
    }
}
__global__ void calculateEnergy(int* lattice, float* energy, float J) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = global_idy * L + global_idx;

    if (global_idx < L && global_idy < L) {
        int siteSpin = lattice[id];
        int rightSpin = (global_idx + 1 < L) ? lattice[id + 1] : lattice[global_idy * L];
        int downSpin = (global_idy + 1 < L) ? lattice[id + L] : lattice[global_idx];
        atomicAdd(energy, -J * siteSpin * (rightSpin + downSpin));
    }
}


// Ising model Metropolis algorithm utilizing shared memory
__global__ void metroIsing(int* lattice, hiprandState* states, float beta, int parity) {
    __shared__ int sharedLattice[THREADS_PER_BLOCK + 2][THREADS_PER_BLOCK + 2];

    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    int local_idx = threadIdx.x + 1;
    int local_idy = threadIdx.y + 1;
    int id = global_idy * L + global_idx;

    if (global_idx < L && global_idy < L) {
        // Load block into shared memory including halo
        sharedLattice[local_idy][local_idx] = lattice[id];

        // Load top halo
        if (threadIdx.y == 0) {
            sharedLattice[0][local_idx] = lattice[id - L];
        }

        // Load bottom halo
        if (threadIdx.y == blockDim.y - 1 || global_idy == L - 1) {
            sharedLattice[THREADS_PER_BLOCK + 1][local_idx] = lattice[id + L];
        }

        // Load left halo
        if (threadIdx.x == 0) {
            sharedLattice[local_idy][0] = lattice[id - 1];
        }

        // Load right halo
        if (threadIdx.x == blockDim.x - 1 || global_idx == L - 1) {
            sharedLattice[local_idy][THREADS_PER_BLOCK + 1] = lattice[id + 1];
        }

        __syncthreads();

        // Apply Metropolis algorithm only to the inner part if parity matches
        if ((global_idx + global_idy) % 2 == parity && global_idx < L && global_idy < L) {
            int siteSpin = sharedLattice[local_idy][local_idx];
            int spinSum = sharedLattice[local_idy + 1][local_idx] + sharedLattice[local_idy - 1][local_idx]
                + sharedLattice[local_idy][local_idx + 1] + sharedLattice[local_idy][local_idx - 1];
            int deltaE = 2 * siteSpin * spinSum;

            // Metropolis acceptance criteria
            if (deltaE <= 0 || hiprand_uniform(&states[id]) < expf(-beta * deltaE)) {
                lattice[id] = -siteSpin;
            }
        }
    }
}

// Main function
int main() {
    int* d_lattice;
    hiprandState* d_states;
    hipMalloc(&d_lattice, N * sizeof(int));
    hipMalloc(&d_states, N * sizeof(hiprandState));
    float* d_energy;
    hipMalloc(&d_energy, sizeof(float));

    dim3 blocks(BLOCKS_PER_GRID, BLOCKS_PER_GRID);
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    // Initialization of lattice to all ones
    int* h_lattice = new int[N];
    for (int i = 0; i < N; ++i) {
        h_lattice[i] = 1;  // Set all spins to 1 (up)
    }
    hipMemcpy(d_lattice, h_lattice, N * sizeof(int), hipMemcpyHostToDevice);

    // Setup RNG states
    setupStates << <blocks, threads >> > (d_states, time(0));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run Monte Carlo simulation for each t from 2 to 0.2
    for (float t = 0.1f; t <= 3.0f; t += 0.1f) {
        float beta = 1.0f / t; // Compute beta as the inverse of temperature t

        hipEventRecord(start);

        // Reset lattice to all ones for each new temperature
        //hipMemcpy(d_lattice, h_lattice, N * sizeof(int), hipMemcpyHostToDevice);

        for (int iter = 0; iter < ITER; ++iter) {
            metroIsing << <blocks, threads >> > (d_lattice, d_states, beta, iter % 2);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Retrieve final lattice state
        hipMemcpy(h_lattice, d_lattice, N * sizeof(int), hipMemcpyDeviceToHost);

        // Compute magnetization
        int mag = 0;
        for (int i = 0; i < N; ++i) {
            mag += h_lattice[i];
        }

        float energy = 0.0f;
        hipMemcpy(d_energy, &energy, sizeof(float), hipMemcpyHostToDevice);

        calculateEnergy << <blocks, threads >> > (d_lattice, d_energy, J);
        hipMemcpy(&energy, d_energy, sizeof(float), hipMemcpyDeviceToHost);

        // Average energy per site
        float energyPerSite = energy / N;

        std::cout << "Temperature: " << t << ", Final magnetization per site: " << static_cast<float>(mag) / N << ", Energy per site: " << energyPerSite << ", Time: " << milliseconds << " ms" << std::endl;
    }


    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete[] h_lattice;
    hipFree(d_lattice);
    hipFree(d_states);

    return 0;
}
