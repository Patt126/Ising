#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>

#define L 1024
#define N (L*L)
#define J 1.00
#define IT 2e9// Number of iterations, should be divisible by 2 for even updates
#define NTHREADS 256 // Number of GPU threads

__device__ int get_index(int row, int col) {
    return (row * L + col) % N;
}

// The lattice uses boolean values, true for spin up (equivalent to 1) and false for spin down (equivalent to -1)
__device__ int delta_energy(bool* lattice, int r, int c) {
    int sum = lattice[get_index((r - 1 + L) % L, c)]
        + lattice[get_index((r + 1) % L, c)]
        + lattice[get_index(r, (c - 1 + L) % L)]
        + lattice[get_index(r, (c + 1) % L)];
    sum = 2 * sum - 4; // Convert sum from [0, 4] to [-4, 4] to match the original spin values
    int spin = lattice[get_index(r, c)] ? 1 : -1; // Convert bool to equivalent spin value
    return 2 * spin * sum;
}

__global__ void flip_spins(bool* lattice, float* prob, float* energy, int* M, hiprandState* states, bool update_black) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    int r = idx / L;
    int c = idx % L;
    bool is_black = ((r + c) % 2 == 0);

    if (is_black == update_black) {
        int delta = delta_energy(lattice, r, c);
        float rnd = hiprand_uniform(&states[idx]);

        if (delta <= 0 || (delta == 4 && rnd < prob[0]) || (delta == 8 && rnd < prob[1])) {
            lattice[get_index(r, c)] = !lattice[get_index(r, c)];
            atomicAdd(energy, delta * J);
            int spin_change = lattice[get_index(r, c)] ? 2 : -2; // Convert bool to equivalent spin change
            atomicAdd(M, spin_change);
        }
    }
}


__global__ void setup_rand_kernel(hiprandState* state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

__global__ void initialize_lattice_kernel(bool* lattice, float* energy, int* M, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float randVal = hiprand_uniform(&states[idx]);
        lattice[idx] = (randVal < 0.5f) ? true : false;

        // Calculate magnetization
        atomicAdd(M, lattice[idx] ? 1 : -1);
    }
}
int main() {
    bool* dev_lattice;
    hipMalloc((void**)&dev_lattice, N * sizeof(bool));

    hiprandState* dev_states;
    hipMalloc((void**)&dev_states, N * sizeof(hiprandState));

    dim3 blocksPerGrid((N + NTHREADS - 1) / NTHREADS, 1, 1);
    dim3 threadsPerBlock(NTHREADS, 1, 1);

    unsigned long seed = static_cast<unsigned long>(time(nullptr));
    setup_rand_kernel << <blocksPerGrid, threadsPerBlock >> > (dev_states, seed);

    float* dev_energy;
    int* dev_M;
    hipMalloc((void**)&dev_energy, sizeof(float));
    hipMalloc((void**)&dev_M, sizeof(int));

    float energy = 0.0f;
    int M = 0;
    hipMemcpy(dev_energy, &energy, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_M, &M, sizeof(int), hipMemcpyHostToDevice);

    initialize_lattice_kernel << <blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_energy, dev_M, dev_states);

    float* dev_probabilities;
    hipMalloc((void**)&dev_probabilities, 2 * sizeof(float));

    for (float T = 0.2f; T <= 3.0f; T += 0.1f) {
        clock_t start_time = clock();

        float prob[2] = { exp(-4 * J / T), exp(-8 * J / T) };
        hipMemcpy(dev_probabilities, prob, 2 * sizeof(float), hipMemcpyHostToDevice);

        // Ensure an even number of iterations for a complete Monte Carlo sweep.
        for (unsigned long i = 0; i < IT / N; i += 2) {
            flip_spins << <blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_probabilities, dev_energy, dev_M, dev_states, true);
            flip_spins << <blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_probabilities, dev_energy, dev_M, dev_states, false);
        }

        hipDeviceSynchronize();

        clock_t end_time = clock();

        double elapsed_secs = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;

        hipMemcpy(&energy, dev_energy, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&M, dev_M, sizeof(int), hipMemcpyDeviceToHost);

        std::cout << "Temperature: " << T << std::endl;
        std::cout << "Final Energy: " << energy / N << std::endl;
        std::cout << "Final Magnetization: " << static_cast<float>(M) / N << std::endl;
        std::cout << "Simulation time (seconds): " << elapsed_secs << std::endl << std::endl;
    }

    hipFree(dev_lattice);
    hipFree(dev_states);
    hipFree(dev_energy);
    hipFree(dev_M);
    hipFree(dev_probabilities);

    return 0;
}
